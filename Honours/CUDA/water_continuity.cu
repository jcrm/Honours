#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>
#include "math.h"
#include "../Source/CUDA/cuda_header.h"

__global__ void cuda_kernel_water_thermo(float *input, float *input_two, float *input_tree, Size size){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){
		if(x_iter +1 < size.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){

					float* water = input_two + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
					float* rain = input_tree + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
					float* thermo = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);

					float qv = water[qv_identifier_];
					float qc = water[qc_identifier_];
					float qr = rain[qr_identifier_];
					float theta =thermo[theta_identifier_];
					float theta_advect = thermo[theta_advect_identifier_];

					float K=beta*qc*qr*z_alt;
					float F=-V*qr/b1*z_alt;
					if(F != 0.f){
						F *= 1.f;
					}
					float A = 0;
					if(qc>aT){
						A=alpha*(qc-aT)*z_alt;
					}
					
					float p=p0*pow((T/T0),(g/R/gamma));
					float TEMP = theta * powf((p0/p),k);
					float est = (es0/p)*exp(a*(TEMP-273)/(TEMP-b));
					float C = g*p/(R*T*pow((p-est),2));
					C += (-a*gamma)*((273-b)/pow((TEMP-b),2))*((1.f/p)+(est/pow((p-est),2)));
					C *= -est*W*epsilon*z_alt;
					
					qv = -C/W;
					qc = (-A-K+C)/W;
					qr = (A+K+F)/W;
					float temp = (latent_heat / (cp * powf(p/p0,k)));
					temp *= C * time_step;
					temp = theta_advect - temp;
					theta = temp;

					water[qv_identifier_] = qv;
					water[qc_identifier_] = qc;
					rain[qr_identifier_] = qr;
					rain[F_identifier_] = F;
					float themp_F = rain[F_identifier_];
					thermo[theta_identifier_] = theta;
					thermo[theta_advect_identifier_] = theta_advect;
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_water_thermo(void *input, void *input_two, void* input_three, Size size){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	//dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_water_thermo<<<Dg,Db>>>((float *)input, (float *)input_two, (float*)input_three, size);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_jacobi() failed to launch error = %d\n", error);
	}
}