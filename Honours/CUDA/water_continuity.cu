#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>
#include "math.h"

#define T0 295.f
#define gamma 6.f/1000.f
#define p0 100000
#define aT 5e-4
#define alpha 1e-3
#define beta 2.f
#define b1 1000.f
#define V 4.f
#define W 8.f
#define g 9.8f
#define R 287.f
#define epsilon 18.02f/29.87f
#define a 17.27f
#define b 35.86f
#define es0 100.f*3.8f
#define z_alt 1000

#define PIXEL_FMT_SIZE 4
#define qv_identifier_ 0
#define qc_identifier_ 1
#define qr_identifier_ 2
#define F_identifier_ 3

__global__ void cuda_kernel_water(unsigned char *input, unsigned char *input_two,float3 size_WHD, size_t pitch, size_t pitch_slice, int pressure_index, int divergence_index){  
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size_WHD.z; ++z_iter){
		if(x_iter +1 < size_WHD.x && x_iter - 1 >= 0){
			if(y_iter + 1 < size_WHD.y && y_iter - 1 >= 0){
				if(z_iter + 1 < size_WHD.z && z_iter - 1 >= 0){
					float qc, qr, qv;
					float K=beta*qc*qr;
					float F=-V*qr/b1;
					float A = 0;
					if(qc>aT){
						A=alpha*(qc-aT);
					}
					float theta;
					float T=T0-gamma*z_alt;
					float p=p0*pow((T/T0),(g/R/gamma));
					float est = (es0/p)*exp(a*(theta-273)/(theta-b));
					float C = g*p/(R*T*pow((p-est),2));
					C += (-a*gamma)*((273-b)/pow((theta-b),2))*((1.f/p)+(est/pow((p-est),2)));
					C *= -est*W*epsilon;
					
					qv = -C/W;
					qc = (-A-K+C)/W;
					qr = (A+K+F)/W;
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_water(void *input, float3 size_WHD, size_t pitch, size_t pitch_slice, int pressure_index, int divergence_index){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size_WHD.x+Db.x-1)/Db.x, (size_WHD.y+Db.y-1)/Db.y);

	cuda_kernel_water<<<Dg,Db>>>((unsigned char *)input,size_WHD, pitch, pitch_slice, pressure_index, divergence_index);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_jacobi() failed to launch error = %d\n", error);
	}
}