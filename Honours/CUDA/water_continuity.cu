#include "hip/hip_runtime.h"
#ifndef _WATER_CONTINUITY_CUDA_
#define _WATER_CONTINUITY_CUDA_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""
#include "math.h"

#include "../Source/CUDA/cuda_header.h"

__global__ void cuda_kernel_water_thermo(float *input, float *input_two, float *input_tree, Size size){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){
		if(x_iter + 1 < size.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){

					float* water = input_two + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
					float* rain = input_tree + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
					float* thermo = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);

					float qv = water[qv_identifier_];
					float qc = water[qc_identifier_];
					float qr = rain[qr_identifier_];
					float theta =thermo[theta_identifier_];
					float theta_advect = thermo[theta_advect_identifier_];

					float K=beta*qc*qr;
					float F=-V*qr/b1;
					float A = 0;
					if(qc>aT){
						A=alpha*(qc-aT);
					}

					float pressure = p0*pow((T/T0),(g/R/gamma));
					float temperature = theta * powf((p0/pressure),k);
					float est = (es0/pressure)*exp(a*(temperature-273)/(temperature-b));
					float pres_minus_est = pow((pressure-est),2);
					float C = (-est*W*epsilon*z_alt) * ((g*pressure/(R*T*pres_minus_est)) + (-a*gamma)*((273.f-b)/pow((temperature-b),2))*((1.f/pressure)+(est/pres_minus_est)));

					qv = (-C/W)*100.f;
					qc = ((-A-K+C)/W)*100.f;
					qr = ((A+K+F)/W)*100.f;

					theta = theta_advect - (latent_heat / (cp * powf(pressure/p0,k))) * C * time_step;

					water[qv_identifier_] = qv;
					water[qc_identifier_] = qc;
					rain[qr_identifier_] = qr;
					rain[F_identifier_] = F;
					thermo[theta_identifier_] = theta;
					thermo[theta_advect_identifier_] = theta_advect;
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_water_thermo(void *input, void *input_two, void* input_three, Size size){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	//dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_water_thermo<<<Dg,Db>>>((float *)input, (float *)input_two, (float*)input_three, size);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_jacobi() failed to launch error = %d\n", error);
	}
}
#endif