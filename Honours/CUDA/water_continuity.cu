#include "hip/hip_runtime.h"
#ifndef _WATER_CONTINUITY_CUDA_
#define _WATER_CONTINUITY_CUDA_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""
#include "math.h"

#include "../Source/CUDA/cuda_header.h"

__global__ void cuda_kernel_water(float *input, float *input_two, float *input_three, Size size, float4 vapor){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; z_iter++){
		float* water = input_two + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
		float* rain = input_three + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
		float* thermo = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);

		float qv = water[qv_identifier_];
		float qc = water[qc_identifier_];
		float qr = rain[qr_identifier_];
		float theta = thermo[theta_identifier_];

		float K=beta*qc*qr;
		float F=-V*qr/b1;
		if(F !=0 ){
			F*=-1;
			F*=-1;
		}
		float A = 0;
		if(qc>aT){
			A=alpha*(qc-aT);
		}
		float temperature = theta * powf((p0/pressure),k);
		float est = (es0/pressure)*expf(a*(temperature-273.f)/(temperature-b));
		float pres_minus_est = powf((pressure-est),2.f);
		float C = (-est*W*epsilon*z_alt) * ((g*1000.f*pressure/(R*T*pres_minus_est)) + (-a*gamma)*((273.f-b)/pow((temperature-b),2))*((1.f/pressure)+(est/pres_minus_est)));
		C *= 1000.f;
		qv = (-C/W);
		qc = ((-A-K+C)/W);
		qr = ((A+K+F)/W);

		water[qv_identifier_] = qv;
		water[qc_identifier_] = qc;
		rain[qr_identifier_] = qr;
		rain[F_identifier_] = F;

		if(x_iter == 0){
			water[qc_identifier_] = 0.f;
			water[qv_identifier_] = vapor.x;
		}else if(x_iter + 1 == size.width_){
			water[qc_identifier_] = 0.f;
			water[qv_identifier_] = vapor.y;
		}else if (y_iter == 0){
			water[qc_identifier_] = 0.f;
			water[qv_identifier_] = 0.0000009f;
		}else if (y_iter + 1 == size.height_){
			water[qc_identifier_] = 0.f;
			water[qv_identifier_] = 0.f;
		}else if (z_iter == 0){
			water[qc_identifier_] = 0.f;
			water[qv_identifier_] = vapor.z;
		}else if (z_iter + 1 == size.depth_){
			water[qc_identifier_] = 0.f;
			water[qv_identifier_] = vapor.w;
		}
	}
}
extern "C"
void cuda_fluid_water(void *input, void *input_two, void* input_three, Size size, float4 vapor){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	//dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_water<<<Dg,Db>>>((float *)input, (float *)input_two, (float*)input_three, size, vapor);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_jacobi() failed to launch error = %d\n", error);
	}
}
#endif