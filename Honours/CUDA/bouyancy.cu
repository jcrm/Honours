#include "hip/hip_runtime.h"
#ifndef _BOUYANCY_CUDA_
#define _BOUYANCY_CUDA_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""

#include "../Source/CUDA/cuda_header.h"

//output velocity derrivitive teture //input velcoity texutre
__global__ void cuda_kernel_bouyancy(float *output, float *input, float *input_two, Size size, Size size_two){ 
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){ 
		if(x_iter +1 < size.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){
					float* output_velocity = output + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
					float* input_thermo = input + (z_iter*size_two.pitch_slice_) + (y_iter*size_two.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
					float* input_water = input_two + (z_iter*size_two.pitch_slice_) + (y_iter*size_two.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
					float theta = input_thermo[theta_identifier_];
					float qv = input_water[qv_identifier_];
					float qh = input_water[qc_identifier_];
					float temp = ((((theta-273)*(1.f+(0.61f*qv))) / (T0-273)) - qh );
					//buoyancy
					float delta = output_velocity[y_identifier_];
					delta += 9.8f * temp * time_step;
					output_velocity[y_identifier_] = delta;
				}
			}
		}
	}
}

extern "C"
void cuda_fluid_bouyancy(void *output, void *input, void *input_two, Size size, Size size_two){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_bouyancy<<<Dg,Db>>>((float *)output, (float *)input, (float *)input_two, size, size_two);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_advect() failed to launch error = %d\n", error);
	}
}
#endif