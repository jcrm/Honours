#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>
#include "../Source/CUDA/cuda_header.h"

__global__ void cuda_kernel_jacobi(unsigned char *pressuredivergence, Size size, int pressure_index, int divergence_index){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;
	for(int i = 0; i < 16; i++){
		if(i%2 == 0){
			pressure_index = 0;
			divergence_index = 1;
		}else{
			pressure_index = 1;
			divergence_index = 0;
		}
		for(z_iter = 0; z_iter < size.depth_; ++z_iter){ 
			if(x_iter +1 < size.width_ && x_iter - 1 >= 0){
				if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
					if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){
					
						unsigned char* cellDivergence = pressuredivergence + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);
						// Get the divergence at the current cell.  
						float dCentre = cellDivergence[divergence_index];

						unsigned char *pLeft = pressuredivergence + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * (x_iter-1));
						unsigned char *pRight = pressuredivergence + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * (x_iter+1));
						unsigned char *pDown = pressuredivergence + (z_iter*size.pitch_slice_) + ((y_iter-1)*size.pitch_) + (PIXEL_FMT_SIZE * x_iter); 
						unsigned char *pUp = pressuredivergence + (z_iter*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE * x_iter); 
						unsigned char *pTop = pressuredivergence + ((z_iter-1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);
						unsigned char *pBottom = pressuredivergence + ((z_iter+1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);

						// Compute the new pressure value for the center cell.
						unsigned char* cellPressure = pressuredivergence + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);
						cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/6.f;
					}
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_jacobi(void *input, Size size, int pressure_index, int divergence_index){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_jacobi<<<Dg,Db>>>((unsigned char *)input, size, pressure_index, divergence_index);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_jacobi() failed to launch error = %d\n", error);
	}
}