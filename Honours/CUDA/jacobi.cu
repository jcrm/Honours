#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

__global__ void cuda_kernel_jacobi(unsigned char *pressuredivergence, float3 size_WHD, size_t pitch, size_t pitch_slice, int pressure_index, int divergence_index){  
	int xIter = blockIdx.x*blockDim.x + threadIdx.x;
	int yIter = blockIdx.y*blockDim.y + threadIdx.y;
	int zIter = 0;

	for(zIter = 0; zIter < size_WHD.z; ++zIter){
		unsigned char* cellDivergence = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*xIter);
		// Get the divergence at the current cell.  
		float dCentre = cellDivergence[divergence_index];
		// Get pressure values from neighboring cells. 
		unsigned char *pLeft, *pRight = NULL;
		unsigned char *pDown, *pUp = NULL;
		unsigned char *pBottom, *pTop = NULL;

		// Compute the new pressure value for the center cell.
		unsigned char* cellPressure = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*xIter);
		/*if((xIter - 1 < 0) && (yIter - 1 < 0) && (zIter - 1 < 0)){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pBottom[pressure_index] + pUp[pressure_index] - dCentre)/3.f;

		}else if((xIter + 1 ==size_WHD.x) && (yIter - 1 < 0) && (zIter - 1 < 0)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pBottom[pressure_index] + pUp[pressure_index] - dCentre)/3.f;

		}else if((xIter - 1 < 0) && (yIter + 1 ==size_WHD.y) && (zIter - 1 < 0)){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pBottom[pressure_index] + pDown[pressure_index] - dCentre)/3.f;

		}else if((xIter + 1 ==size_WHD.x) && (yIter + 1 ==size_WHD.y) && (zIter - 1 < 0)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pBottom[pressure_index] + pDown[pressure_index] - dCentre)/3.f;

		}else if((xIter - 1 < 0) && (yIter - 1 < 0) && (zIter + 1 ==size_WHD.y)){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pTop[pressure_index] + pUp[pressure_index] - dCentre)/3.f;

		}else if((xIter + 1 ==size_WHD.x) && (yIter - 1 < 0) && (zIter + 1 ==size_WHD.y)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pTop[pressure_index] + pUp[pressure_index] - dCentre)/3.f;

		}else if((xIter - 1 < 0) && (yIter + 1 ==size_WHD.y) && (zIter + 1 ==size_WHD.y)){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pTop[pressure_index] + pDown[pressure_index] - dCentre)/3.f;

		}else if((xIter + 1 ==size_WHD.x) && (yIter + 1 ==size_WHD.y) && (zIter + 1 ==size_WHD.y)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pTop[pressure_index] + pDown[pressure_index] - dCentre)/3.f;

		}else if((xIter - 1 < 0) && (yIter - 1 < 0)){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pUp[pressure_index] - dCentre)/4.f;

		}else if((xIter + 1 ==size_WHD.x) && (yIter - 1 < 0)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pUp[pressure_index] - dCentre)/4.f;

		}else if((xIter - 1 < 0) && (yIter + 1 ==size_WHD.y)){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pDown[pressure_index] - dCentre)/4.f;

		}else if((xIter + 1 ==size_WHD.x) && (yIter + 1 ==size_WHD.y)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pDown[pressure_index] - dCentre)/4.f;

		}else if((xIter - 1 < 0) && (zIter - 1 < 0)){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pBottom[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/4.f;

		}else if((xIter + 1 ==size_WHD.x) && (zIter - 1 < 0)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pBottom[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/4.f;

		}else if((xIter - 1 < 0) && (zIter + 1 ==size_WHD.y)){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pTop[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/4.f;

		}else if((xIter + 1 ==size_WHD.x) && (zIter + 1 ==size_WHD.y)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pTop[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/4.f;

		}else if((yIter - 1 < 0) && (zIter - 1 < 0)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pBottom[pressure_index] + pUp[pressure_index] - dCentre)/4.f;

		}else if((yIter + 1 ==size_WHD.y) && (zIter - 1 < 0)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pBottom[pressure_index] + pDown[pressure_index] - dCentre)/4.f;

		}else if((yIter - 1 < 0) && (zIter + 1 ==size_WHD.y)){
			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index]+ pTop[pressure_index] + pUp[pressure_index] - dCentre)/4.f;
		}else if((yIter + 1 ==size_WHD.y) && (zIter + 1 ==size_WHD.y)){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pTop[pressure_index] + pDown[pressure_index] - dCentre)/4.f;

		}else if(xIter - 1 < 0){

			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pRight[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/5.f;

		}else if(xIter + 1 ==size_WHD.x){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/5.f;

		}else if(yIter - 1 < 0){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pUp[pressure_index] - dCentre)/5.f;

		}else if(yIter + 1 ==size_WHD.y){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pDown[pressure_index] - dCentre)/5.f;

		}else if(zIter - 1 < 0){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pBottom[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/5.f;

		}else if(zIter + 1 ==size_WHD.y){

			pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
			pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
			pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
			pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
			pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
			cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pTop[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/5.f;

		}else{*/
		if(xIter +1 < size_WHD.x){
			if(xIter - 1 > 0){
				if(yIter + 1 < size_WHD.y){
					if(yIter - 1 > 0){
						if(zIter + 1 < size_WHD.z){
							if(zIter - 1 > 0){
								pLeft = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
								pRight = pressuredivergence + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
								pDown = pressuredivergence + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
								pUp = pressuredivergence + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
								pTop = pressuredivergence + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
								pBottom = pressuredivergence + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
								cellPressure[pressure_index] = (pLeft[pressure_index] + pRight[pressure_index] + pBottom[pressure_index] + pTop[pressure_index] + pUp[pressure_index] + pDown[pressure_index] - dCentre)/6.f;
							}
						}
					}
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_jacobi(void *pressuredivergence, float3 size_WHD, size_t pitch, size_t pitch_slice, int pressure_index, int divergence_index){
	hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((size_WHD.x+Db.x-1)/Db.x, (size_WHD.y+Db.y-1)/Db.y);

    cuda_kernel_jacobi<<<Dg,Db>>>((unsigned char *)pressuredivergence,size_WHD, pitch, pitch_slice, pressure_index, divergence_index);

    error = hipGetLastError();
    if (error != hipSuccess){
        printf("cuda_kernel_jacobi() failed to launch error = %d\n", error);
    }
}