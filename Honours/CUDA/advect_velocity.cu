#include "hip/hip_runtime.h"
#ifndef ADVECT_VELOCITY_CUDA_
#define ADVECT_VELOCITY_CUDA_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""

#include "../Source/CUDA/cuda_header.h"

//output velocity derrivitive teture //input velcoity texutre
__global__ void cuda_kernel_advect_velocity(float *output, float*input, Size size){ 
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){ 
		if(x_iter +1 < size.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){
					float *fieldRight = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * (x_iter+1));
					float *fieldDown = input + (z_iter*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter); 
					float*fieldRightCorner = input + (z_iter*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * (x_iter+1));
					float *field = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);

					float *fieldRightBack = input + ((z_iter+1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * (x_iter+1));
					float *fieldDownBack = input + ((z_iter+1)*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter); 
					float *fieldRightCornerBack = input + ((z_iter+1)*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * (x_iter+1));
					float *fieldBack = input + ((z_iter+1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);

					float temp_X_1 = field[x_identifier_] +((fieldRight[x_identifier_]-field[x_identifier_])*0.5f);
					float temp_Y_1 = field[y_identifier_] +((fieldRight[y_identifier_]-field[y_identifier_])*0.5f);
					float temp_Z_1 = field[z_identifier_] +((fieldRight[z_identifier_]-field[z_identifier_])*0.5f);

					float temp_X_2 = fieldDown[x_identifier_] +((fieldRightCorner[x_identifier_]-fieldDown[x_identifier_])*0.5f);
					float temp_Y_2 = fieldDown[y_identifier_] +((fieldRightCorner[y_identifier_]-fieldDown[y_identifier_])*0.5f);
					float temp_Z_2 = fieldDown[z_identifier_] +((fieldRightCorner[z_identifier_]-fieldDown[z_identifier_])*0.5f);

					float temp_X_3 = fieldBack[x_identifier_] +((fieldRightBack[x_identifier_]-fieldBack[x_identifier_])*0.5f);
					float temp_Y_3 = fieldBack[y_identifier_] +((fieldRightBack[y_identifier_]-fieldBack[y_identifier_])*0.5f);
					float temp_Z_3 = fieldBack[z_identifier_] +((fieldRightBack[z_identifier_]-fieldBack[z_identifier_])*0.5f);

					float temp_X_4 = fieldDownBack[x_identifier_] +((fieldRightCornerBack[x_identifier_]-fieldDownBack[x_identifier_])*0.5f);
					float temp_Y_4 = fieldDownBack[y_identifier_] +((fieldRightCornerBack[y_identifier_]-fieldDownBack[y_identifier_])*0.5f);
					float temp_Z_4 = fieldDownBack[z_identifier_] +((fieldRightCornerBack[z_identifier_]-fieldDownBack[z_identifier_])*0.5f);

					temp_X_1 =(temp_X_1 + (temp_X_2-temp_X_1)*0.5f);
					temp_Y_1 =(temp_Y_1 + (temp_Y_2-temp_Y_1)*0.5f);
					temp_Z_1 =(temp_Z_1 + (temp_Z_2-temp_Z_1)*0.5f);

					temp_X_3 =(temp_X_3 + (temp_X_4-temp_X_3)*0.5f);
					temp_Y_3 =(temp_Y_3 + (temp_Y_4-temp_Y_3)*0.5f);
					temp_Z_3 =(temp_Z_3 + (temp_Z_4-temp_Z_3)*0.5f);

					float*output_velocity = output + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
					output_velocity[x_identifier_] = temp_X_1 + ((temp_X_3-temp_X_1)*0.5f);
					output_velocity[y_identifier_] = temp_Y_1 + ((temp_Y_3-temp_Y_1)*0.5f);
					output_velocity[z_identifier_] = temp_Z_1 + ((temp_Z_3-temp_Z_1)*0.5f);
				}
			}
		}
		
	}
	
}

extern "C"
void cuda_fluid_advect_velocity(void *output, void *input, Size size){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_advect_velocity<<<Dg,Db>>>((float *)output, (float *)input, size);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_advect() failed to launch error = %d\n", error);
	}
}
#endif