#include "hip/hip_runtime.h"
#ifndef ADVECT_VELOCITY_CUDA_
#define ADVECT_VELOCITY_CUDA_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""

#include "../Source/CUDA/cuda_header.h"

//output velocity derrivitive teture //input velcoity texutre
__global__ void cuda_kernel_advect_velocity(float *output, float*input, Size size){ 
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; z_iter++){ 
		if((x_iter - 1 >= 0 && x_iter + 1 < size.width_) && (y_iter - 1 >= 0 && y_iter + 1 < size.height_) && (z_iter - 1 >= 0 && z_iter + 1 < size.depth_)){
			float *cellVelocity = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
			float3 pos = {x_iter, y_iter, z_iter};
			float3 cell_velocity = {cellVelocity[x_identifier_], cellVelocity[y_identifier_], cellVelocity[z_identifier_]};
			pos.x = pos.x - (time_step * cell_velocity.x);
			pos.y = pos.y - (time_step * cell_velocity.y);
			pos.z = pos.z - (time_step * cell_velocity.z);
			
			int3 location = {pos.x,pos.y, pos.z};
			if(location.x < 0){
				location.x = 0;
			}
			if(location.y <0){
				location.y = 0;
			}
			if(location.z < 0){
				location.z = 0;
			}
			if(location.x >= size.width_){
				location.x = size.width_ - 1;
			}
			if(location.y >= size.height_){
				location.y = size.height_ - 1;
			}
			if(location.z >= size.depth_){
				location.z = size.depth_ - 1;
			}
			int3 location_two = {location.x+1,location.y+1, location.z+1};
			if(location_two.x >= size.width_){
				location_two.x = size.width_ - 1;
			}
			if(location_two.y >= size.height_){
				location_two.y = size.height_ - 1;
			}
			if(location_two.z >= size.depth_){
				location_two.z = size.depth_ - 1;
			}
			float *field_left_up = input + (location.z*size.pitch_slice_) + (location.y*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * location.x);
			float *field_left_down = input + (location.z*size.pitch_slice_) + (location_two.y*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * location.x);
			float *field_right_up = input + (location.z*size.pitch_slice_) + (location.y*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * location_two.x);
			float *field_right_down = input + (location.z*size.pitch_slice_) + (location_two.y*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * location_two.x);

			float *field_left_up_back = input + (location_two.z*size.pitch_slice_) + (location.y*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * location.x);
			float *field_left_down_back = input + (location_two.z*size.pitch_slice_) + (location_two.y*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * location.x);
			float *field_right_up_back = input + (location_two.z*size.pitch_slice_) + (location.y*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * location_two.x);
			float *field_right_down_back = input + (location_two.z*size.pitch_slice_) + (location_two.y*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * location_two.x);

			float temp_X_1 = field_left_up[x_identifier_] + field_left_down[x_identifier_] + field_right_up[x_identifier_] + field_right_down[x_identifier_];
			float temp_Y_1 = field_left_up[y_identifier_] + field_left_down[y_identifier_] + field_right_up[y_identifier_] + field_right_down[y_identifier_];
			float temp_Z_1 = field_left_up[z_identifier_] + field_left_down[z_identifier_] + field_right_up[z_identifier_] + field_right_down[z_identifier_];

			float temp_X_2 = field_left_up_back[x_identifier_] + field_left_down_back[x_identifier_] + field_right_up_back[x_identifier_] + field_right_down_back[x_identifier_];
			float temp_Y_2 = field_left_up_back[y_identifier_] + field_left_down_back[y_identifier_] + field_right_up_back[y_identifier_] + field_right_down_back[x_identifier_];
			float temp_Z_2 = field_left_up_back[z_identifier_] + field_left_down_back[z_identifier_] + field_right_up_back[z_identifier_] + field_right_down_back[x_identifier_];


			temp_X_1 /=4.f;
			temp_Y_1 /=4.f;
			temp_Z_1 /=4.f;

			temp_X_2 /=4.f;
			temp_Y_2 /=4.f;
			temp_Z_2 /=4.f;
			
			float*output_velocity = output + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
			output_velocity[x_identifier_] = (temp_X_1 + temp_X_2)/2.f;
			output_velocity[y_identifier_] = (temp_Y_1 + temp_Y_2)/2.f;
			output_velocity[z_identifier_] = (temp_Z_1 + temp_Z_2)/2.f;
		}
	}
	
}

extern "C"
void cuda_fluid_advect_velocity(void *output, void *input, Size size){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_advect_velocity<<<Dg,Db>>>((float *)output, (float *)input, size);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_advect() failed to launch error = %d\n", error);
	}
}
#endif