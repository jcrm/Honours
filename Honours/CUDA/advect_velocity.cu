#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>
#include "../Source/CUDA/cuda_header.h"

#define PIXEL_FMT_SIZE 4
#define timeStep 1.f
#define x_identifier_ 0
#define y_identifier_ 1
#define z_identifier_ 2
//output velocity derrivitive teture //input velcoity texutre
__global__ void cuda_kernel_advect_velocity(unsigned char *output, unsigned char *input, Size size){ 
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){ 
		if(x_iter +1 < size.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){
					unsigned char *fieldRight = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * (x_iter+1));
					unsigned char *fieldDown = input + (z_iter*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE * x_iter); 
					unsigned char *fieldRightCorner = input + (z_iter*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE * (x_iter+1));
					unsigned char *field = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);

					unsigned char *fieldRightBack = input + ((z_iter+1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * (x_iter+1));
					unsigned char *fieldDownBack = input + ((z_iter+1)*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE * x_iter); 
					unsigned char *fieldRightCornerBack = input + ((z_iter+1)*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE * (x_iter+1));
					unsigned char *fieldBack = input + ((z_iter+1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);

					float temp_X_1 = field[x_identifier_] +((fieldRight[x_identifier_]-field[x_identifier_])*0.5f);
					float temp_Y_1 = field[y_identifier_] +((fieldRight[y_identifier_]-field[y_identifier_])*0.5f);
					float temp_Z_1 = field[z_identifier_] +((fieldRight[z_identifier_]-field[z_identifier_])*0.5f);

					float temp_X_2 = fieldDown[x_identifier_] +((fieldRightCorner[x_identifier_]-fieldDown[x_identifier_])*0.5f);
					float temp_Y_2 = fieldDown[y_identifier_] +((fieldRightCorner[y_identifier_]-fieldDown[y_identifier_])*0.5f);
					float temp_Z_2 = fieldDown[z_identifier_] +((fieldRightCorner[z_identifier_]-fieldDown[z_identifier_])*0.5f);

					float temp_X_3 = fieldBack[x_identifier_] +((fieldRightBack[x_identifier_]-fieldBack[x_identifier_])*0.5f);
					float temp_Y_3 = fieldBack[y_identifier_] +((fieldRightBack[y_identifier_]-fieldBack[y_identifier_])*0.5f);
					float temp_Z_3 = fieldBack[z_identifier_] +((fieldRightBack[z_identifier_]-fieldBack[z_identifier_])*0.5f);

					float temp_X_4 = fieldDownBack[x_identifier_] +((fieldRightCornerBack[x_identifier_]-fieldDownBack[x_identifier_])*0.5f);
					float temp_Y_4 = fieldDownBack[y_identifier_] +((fieldRightCornerBack[y_identifier_]-fieldDownBack[y_identifier_])*0.5f);
					float temp_Z_4 = fieldDownBack[z_identifier_] +((fieldRightCornerBack[z_identifier_]-fieldDownBack[z_identifier_])*0.5f);

					temp_X_1 =(temp_X_1 + (temp_X_2-temp_X_1)*0.5f);
					temp_Y_1 =(temp_Y_1 + (temp_Y_2-temp_Y_1)*0.5f);
					temp_Z_1 =(temp_Z_1 + (temp_Z_2-temp_Z_1)*0.5f);

					temp_X_3 =(temp_X_3 + (temp_X_4-temp_X_3)*0.5f);
					temp_Y_3 =(temp_Y_3 + (temp_Y_4-temp_Y_3)*0.5f);
					temp_Z_3 =(temp_Z_3 + (temp_Z_4-temp_Z_3)*0.5f);

					unsigned char *output_velocity = output + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);
					output_velocity[x_identifier_] = signed int(temp_X_1 + ((temp_X_3-temp_X_1)*0.5f));
					output_velocity[y_identifier_] = signed int(temp_Y_1 + ((temp_Y_3-temp_Y_1)*0.5f));
					output_velocity[z_identifier_] = signed int(temp_Z_1 + ((temp_Z_3-temp_Z_1)*0.5f));
				}
			}
		}
		
	}
	
}

extern "C"
void cuda_fluid_advect_velocity(void *output, void *input, Size size){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_advect_velocity<<<Dg,Db>>>((unsigned char *)output, (unsigned char *)input, size);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_advect() failed to launch error = %d\n", error);
	}
}