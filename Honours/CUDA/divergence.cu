#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

__global__ void cuda_kernel_divergence(unsigned char* divergence, unsigned char* velocityInput, size_t pitch, size_t pitchSlice){
	int xIter = blockIdx.x*blockDim.x + threadIdx.x;
	int yIter = blockIdx.y*blockDim.y + threadIdx.y;
	int zIter = 0;

	// Get velocity values from neighboring cells.  
	unsigned char *fieldLeft = velocityInput + (zIter*pitchSlice) + (yIter*pitch) + (4*(xIter-1));
	unsigned char *fieldRight = velocityInput + (zIter*pitchSlice) + (yIter*pitch) + (4*(xIter+1));
	unsigned char *fieldBottom = velocityInput + ((zIter+1)*pitchSlice) + (yIter*pitch) + (4*xIter);  
	unsigned char *fieldTop = velocityInput + ((zIter-1)*pitchSlice) + (yIter*pitch) + (4*xIter);   
	unsigned char *fieldDown = velocityInput + (zIter*pitchSlice) + ((yIter-1)*pitch) + (4*xIter); 
	unsigned char *fieldUp = velocityInput + (zIter*pitchSlice) + ((yIter+1)*pitch) + (4*xIter);
	// Compute the velocity's divergence using central differences.  
	divergence[0] =  0.5 * ((fieldRight[0] - fieldLeft[0])+  
								(fieldTop[1] - fieldBottom[1])+  
								(fieldUp[2] - fieldDown[2]));  
}
extern "C"
void cuda_fluid_divergence(unsigned char* divergence, unsigned char* velocityInput, float3 sizeWHD, size_t pitch, size_t pitchSlice){
	hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((sizeWHD.x+Db.x-1)/Db.x, (sizeWHD.y+Db.y-1)/Db.y);

    cuda_kernel_divergence<<<Dg,Db>>>((unsigned char *)divergence, (unsigned char *)velocityInput, pitch, pitchSlice);

    error = hipGetLastError();
    if (error != hipSuccess){
        printf("cuda_kernel_texture_3d() failed to launch error = %d\n", error);
    }
}
