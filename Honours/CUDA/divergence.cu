#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>
#include "../Source/CUDA/cuda_header.h"

//output diverrgnece texture //input velocity derrivitive teture
__global__ void cuda_kernel_divergence(unsigned char* output, unsigned char* input, Size size, int divergence_index){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){ 
		if(x_iter +1 < size.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){
					unsigned char *fieldLeft = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * (x_iter-1));
					unsigned char *fieldRight = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * (x_iter+1));
					unsigned char *fieldDown = input + (z_iter*size.pitch_slice_) + ((y_iter-1)*size.pitch_) + (PIXEL_FMT_SIZE * x_iter); 
					unsigned char *fieldUp = input + (z_iter*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE * x_iter); 
					unsigned char *fieldTop = input + ((z_iter-1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);
					unsigned char *fieldBottom = input + ((z_iter+1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);
					unsigned char *output_divergence = output + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE * x_iter);
					output_divergence[divergence_index] = signed int(0.5f * ((signed int(fieldRight[x_identifier_]) - signed int(fieldLeft[x_identifier_])) + 
						(signed int(fieldTop[y_identifier_]) - signed int(fieldBottom[y_identifier_])) + 
						(signed int(fieldUp[z_identifier_]) - signed int(fieldDown[z_identifier_]))));
					// Compute the velocity's divergence using central differences.  
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_divergence(void *divergence, void *input, Size size, int divergence_index){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_divergence<<<Dg,Db>>>((unsigned char *)divergence, (unsigned char *)input, size, divergence_index);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_divergence() failed to launch error = %d\n", error);
	}
}
