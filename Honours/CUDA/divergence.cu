#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

__global__ void cuda_kernel_divergence(unsigned char* divergence, unsigned char* velocityInput,float3 sizeWHD, size_t pitch, size_t pitchSlice, int divergenceIndex){
	int xIter = blockIdx.x*blockDim.x + threadIdx.x;
	int yIter = blockIdx.y*blockDim.y + threadIdx.y;
	int zIter = 0;

	for(zIter = 0; zIter < sizeWHD.z; ++zIter){ 
		// Get velocity values from neighboring cells.  
		unsigned char *fieldLeft = velocityInput + (zIter*pitchSlice) + (yIter*pitch) + (4*(xIter-1));
		unsigned char *fieldRight = velocityInput + (zIter*pitchSlice) + (yIter*pitch) + (4*(xIter+1));

		unsigned char *fieldUp = velocityInput + ((zIter+1)*pitchSlice) + (yIter*pitch) + (4*xIter);
		unsigned char *fieldDown = velocityInput + ((zIter-1)*pitchSlice) + (yIter*pitch) + (4*xIter);

		unsigned char *fieldTop = velocityInput + (zIter*pitchSlice) + ((yIter-1)*pitch) + (4*xIter); 
		unsigned char *fieldBottom = velocityInput + (zIter*pitchSlice) + ((yIter+1)*pitch) + (4*xIter);

		unsigned char* cellDivergence = divergence + (zIter*pitchSlice) + (yIter*pitch) + (4*xIter);
		// Compute the velocity's divergence using central differences.  
		cellDivergence[divergenceIndex] =  0.5 * ((fieldRight[0] - fieldLeft[0])+  
				(fieldTop[1] - fieldBottom[1]) + (fieldUp[2] - fieldDown[2])); 
	}
}
extern "C"
void cuda_fluid_divergence(void *divergence, void *velocityInput, float3 sizeWHD, size_t pitch, size_t pitchSlice, int divergenceIndex){
	hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((sizeWHD.x+Db.x-1)/Db.x, (sizeWHD.y+Db.y-1)/Db.y);

    cuda_kernel_divergence<<<Dg,Db>>>((unsigned char *)divergence, (unsigned char *)velocityInput,sizeWHD, pitch, pitchSlice, divergenceIndex);

    error = hipGetLastError();
    if (error != hipSuccess){
        printf("cuda_kernel_divergence() failed to launch error = %d\n", error);
    }
}
