#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>
//output diverrgnece texture //input velocity derrivitive teture
__global__ void cuda_kernel_divergence(unsigned char* output, unsigned char* input,float3 size_WHD, size_t pitch, size_t pitch_slice, int divergence_index){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size_WHD.z; ++z_iter){ 
		if(x_iter +1 < size_WHD.x && x_iter - 1 > 0){
			if(y_iter + 1 < size_WHD.y && y_iter - 1 > 0){
				if(z_iter + 1 < size_WHD.z && z_iter - 1 > 0){
					unsigned char *fieldLeft = input + (z_iter*pitch_slice) + (y_iter*pitch) + (4*(x_iter-1));
					unsigned char *fieldRight = input + (z_iter*pitch_slice) + (y_iter*pitch) + (4*(x_iter+1));
					unsigned char *fieldDown = input + (z_iter*pitch_slice) + ((y_iter-1)*pitch) + (4*x_iter); 
					unsigned char *fieldUp = input + (z_iter*pitch_slice) + ((y_iter+1)*pitch) + (4*x_iter); 
					unsigned char *fieldTop = input + ((z_iter-1)*pitch_slice) + (y_iter*pitch) + (4*x_iter);
					unsigned char *fieldBottom = input + ((z_iter+1)*pitch_slice) + (y_iter*pitch) + (4*x_iter);
					unsigned char *output_divergence = output + (z_iter*pitch_slice) + (y_iter*pitch) + (4*x_iter);
					output_divergence[divergence_index] = signed int(0.5f * ((signed int(fieldRight[0]) - signed int(fieldLeft[0])) + 
						(signed int(fieldTop[1]) - signed int(fieldBottom[1])) + (signed int(fieldUp[2]) - signed int(fieldDown[2]))));
					// Compute the velocity's divergence using central differences.  
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_divergence(void *divergence, void *input, float3 size_WHD, size_t pitch, size_t pitch_slice, int divergence_index){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size_WHD.x+Db.x-1)/Db.x, (size_WHD.y+Db.y-1)/Db.y);

	cuda_kernel_divergence<<<Dg,Db>>>((unsigned char *)divergence, (unsigned char *)input,size_WHD, pitch, pitch_slice, divergence_index);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_divergence() failed to launch error = %d\n", error);
	}
}
