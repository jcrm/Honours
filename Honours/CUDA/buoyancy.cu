#pragma once
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include "math.h"

#define PIXEL_FMT_SIZE 4
#define qv_identifier_ 0
#define qc_identifier_ 1
#define qr_identifier_ 2
#define F_identifier_ 3

__global__ void cuda_kernel_bouyancy(unsigned char *input, float3 size_WHD, size_t pitch, size_t pitch_slice, int pressure_index, int divergence_index){  
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size_WHD.z; ++z_iter){
		if(x_iter +1 < size_WHD.x && x_iter - 1 >= 0){
			if(y_iter + 1 < size_WHD.y && y_iter - 1 >= 0){
				if(z_iter + 1 < size_WHD.z && z_iter - 1 >= 0){

				}
			}
		}
	}
}
extern "C"
void cuda_fluid_bouyancy(void *input, float3 size_WHD, size_t pitch, size_t pitch_slice, int pressure_index, int divergence_index){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size_WHD.x+Db.x-1)/Db.x, (size_WHD.y+Db.y-1)/Db.y);

	cuda_kernel_bouyancy<<<Dg,Db>>>((unsigned char *)input,size_WHD, pitch, pitch_slice, pressure_index, divergence_index);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_jacobi() failed to launch error = %d\n", error);
	}
}