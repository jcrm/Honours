#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>

#define timeStep 1.f
//output velocity derrivitive teture //input velcoity texutre
__global__ void cuda_kernel_advect_two_texture(unsigned char *output, unsigned char *input, float3 size_WHD, size_t pitch, size_t pitch_slice){ 
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size_WHD.z; ++z_iter){ 
		if(x_iter +1 < size_WHD.x && x_iter - 1 > 0){
			if(y_iter + 1 < size_WHD.y && y_iter - 1 > 0){
				if(z_iter + 1 < size_WHD.z && z_iter - 1 > 0){
					unsigned char *fieldRight = input + (z_iter*pitch_slice) + (y_iter*pitch) + (4*(x_iter+1));
					unsigned char *fieldDown = input + (z_iter*pitch_slice) + ((y_iter+1)*pitch) + (4*x_iter); 
					unsigned char *fieldRightCorner = input + (z_iter*pitch_slice) + ((y_iter+1)*pitch) + (4*(x_iter+1));
					unsigned char *field = input + (z_iter*pitch_slice) + (y_iter*pitch) + (4*x_iter);

					unsigned char *fieldRightBack = input + ((z_iter+1)*pitch_slice) + (y_iter*pitch) + (4*(x_iter+1));
					unsigned char *fieldDownBack = input + ((z_iter+1)*pitch_slice) + ((y_iter+1)*pitch) + (4*x_iter); 
					unsigned char *fieldRightCornerBack = input + ((z_iter+1)*pitch_slice) + ((y_iter+1)*pitch) + (4*(x_iter+1));
					unsigned char *fieldBack = input + ((z_iter+1)*pitch_slice) + (y_iter*pitch) + (4*x_iter);

					float temp_X_1 = field[0] +((fieldRight[0]-field[0])*0.5f);
					float temp_Y_1 = field[1] +((fieldRight[1]-field[1])*0.5f);
					float temp_Z_1 = field[2] +((fieldRight[2]-field[2])*0.5f);

					float temp_X_2 = fieldDown[0] +((fieldRightCorner[0]-fieldDown[0])*0.5f);
					float temp_Y_2 = fieldDown[1] +((fieldRightCorner[1]-fieldDown[1])*0.5f);
					float temp_Z_2 = fieldDown[2] +((fieldRightCorner[2]-fieldDown[2])*0.5f);

					float temp_X_3 = fieldBack[0] +((fieldRightBack[0]-fieldBack[0])*0.5f);
					float temp_Y_3 = fieldBack[1] +((fieldRightBack[1]-fieldBack[1])*0.5f);
					float temp_Z_3 = fieldBack[2] +((fieldRightBack[2]-fieldBack[2])*0.5f);

					float temp_X_4 = fieldDownBack[0] +((fieldRightCornerBack[0]-fieldDownBack[0])*0.5f);
					float temp_Y_4 = fieldDownBack[1] +((fieldRightCornerBack[1]-fieldDownBack[1])*0.5f);
					float temp_Z_4 = fieldDownBack[2] +((fieldRightCornerBack[2]-fieldDownBack[2])*0.5f);

					temp_X_1 =(temp_X_1 + (temp_X_2-temp_X_1)*0.5f);
					temp_Y_1 =(temp_Y_1 + (temp_Y_2-temp_Y_1)*0.5f);
					temp_Z_1 =(temp_Z_1 + (temp_Z_2-temp_Z_1)*0.5f);

					temp_X_3 =(temp_X_3 + (temp_X_4-temp_X_3)*0.5f);
					temp_Y_3 =(temp_Y_3 + (temp_Y_4-temp_Y_3)*0.5f);
					temp_Z_3 =(temp_Z_3 + (temp_Z_4-temp_Z_3)*0.5f);

					unsigned char *output_velocity = output + (z_iter*pitch_slice) + (y_iter*pitch) + (4*x_iter);
					output_velocity[0] = signed int(temp_X_1 + ((temp_X_3-temp_X_1)*0.5f));
					output_velocity[1] = signed int(temp_Y_1 + ((temp_Y_3-temp_Y_1)*0.5f));
					output_velocity[2] = signed int(temp_Z_1 + ((temp_Z_3-temp_Z_1)*0.5f));
				}
			}
		}
	}
}

extern "C"
void cuda_fluid_advect_two_texture(void *output, void *input, float3 size_WHD, size_t pitch, size_t pitch_slice){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size_WHD.x+Db.x-1)/Db.x, (size_WHD.y+Db.y-1)/Db.y);

	cuda_kernel_advect_two_texture<<<Dg,Db>>>((unsigned char *)output, (unsigned char *)input, size_WHD, pitch, pitch_slice);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_advect() failed to launch error = %d\n", error);
	}
}