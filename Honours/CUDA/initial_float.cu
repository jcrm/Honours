#include "hip/hip_runtime.h"
#ifndef _INITIAL_FLOAT_
#define _INITIAL_FLOAT_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""

#include "../Source/CUDA/cuda_header.h"


__global__ void cuda_kernel_initial_float(float *input, Size size, float value){ 
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){ 
		//location is z slide + y position + variable size time x position
		float *cell_value = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
		cell_value[0] = value;
		cell_value[1] = 0.f;
	}
}
extern "C"
void cuda_fluid_initial_float(void *input, Size size, float value){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_initial_float<<<Dg,Db>>>((float*)input, size, value);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_initial() failed to launch error = %d\n", error);
	}
}
#endif