#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>
#include "math.h"
#include "../Source/CUDA/cuda_header.h"

__global__ void cuda_kernel_rain(float *output, float *input, Size size, Size size_two){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;
	float rain_sum = 0.f;
	int ident = 0;
	for(z_iter = 0; z_iter < size_two.depth_; ++z_iter){
		if(x_iter +1 < size_two.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size_two.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size_two.depth_ && z_iter - 1 >= 0){
					float* cell_rain = input + (z_iter*size_two.pitch_slice_) + (y_iter*size_two.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
					rain_sum += cell_rain[F_identifier_];
				}
			}
		}
	}
	int xIter = x_iter;
	int yIter = y_iter;
	if(x_iter%2 != 0 && y_iter%2 != 0){
		xIter--;
		yIter--;
		ident = 3;
	}else if(x_iter%2 != 0){
		xIter--;
		ident = 1;
	}else if(y_iter%2 != 0){
		yIter--;
		ident = 2;
	}
	xIter /= 2;
	yIter /= 2;
	float* rain = output + (yIter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * xIter);
	rain[ident] = rain_sum;
}
extern "C"
void cuda_fluid_rain(void *output, void *input, Size size, Size size_two){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	//dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);
	dim3 Dg = dim3((size_two.width_+Db.x-1)/Db.x, (size_two.height_+Db.y-1)/Db.y);

	cuda_kernel_rain<<<Dg,Db>>>((float *)output, (float *)input, size, size_two);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_jacobi() failed to launch error = %d\n", error);
	}
}