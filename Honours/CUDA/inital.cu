#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include ""
#include <hip/hip_runtime.h>

__global__ void cuda_kernel_initial(unsigned char *input, float3 size_WHD, size_t pitch, size_t pitch_slice, float value){ 
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size_WHD.z; ++z_iter){ 
		//location is z slide + y position + variable size time x position
		unsigned char *cell_value = input + (z_iter*pitch_slice) + (y_iter*pitch) + (4*x_iter);
		cell_value[0] = signed int(value);
		cell_value[1] = signed int(0);
		cell_value[2] = signed int(0);
		cell_value[3] = signed int(0);
		if(x_iter +1 < size_WHD.x && x_iter - 1 > 0){
			if(y_iter + 1 < size_WHD.y && y_iter - 1 > 0){
				if(z_iter + 1 < size_WHD.z && z_iter - 1 > 0){
					cell_value[0] = signed int(0);
					cell_value[1] = signed int(0);
					cell_value[2] = signed int(0);
					cell_value[3] = signed int(0);
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_initial(void *velocityinput, float3 size_WHD, size_t pitch, size_t pitch_slice, float value){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size_WHD.x+Db.x-1)/Db.x, (size_WHD.y+Db.y-1)/Db.y);

	cuda_kernel_initial<<<Dg,Db>>>((unsigned char *)velocityinput, size_WHD, pitch, pitch_slice, value);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_initial() failed to launch error = %d\n", error);
	}
}