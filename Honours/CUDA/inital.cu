#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

__global__ void cuda_kernel_initial(unsigned char *velocityInput, float3 size_WHD, size_t pitch, size_t pitch_slice, float value){ 
	int xIter = blockIdx.x*blockDim.x + threadIdx.x;
	int yIter = blockIdx.y*blockDim.y + threadIdx.y;
	int zIter = 0;

	for(zIter = 0; zIter < size_WHD.z; ++zIter){ 
		if(xIter > ((size_WHD.x/2) - 10) && xIter < ((size_WHD.x/2) + 10)){
			if(yIter > ((size_WHD.y/2) - 10) && yIter < ((size_WHD.y/2) + 10)){
				if(zIter > ((size_WHD.z/2) - 10) && zIter < ((size_WHD.z/2) + 10)){
					//location is z slide + y position + variable size time x position
					unsigned char* cellVelocity = velocityInput + (zIter*pitch_slice) + (yIter*pitch) + (4*xIter);
					cellVelocity[0] = value;
					cellVelocity[1] = value;
					cellVelocity[2] = value;
					cellVelocity[3] = 1.0f;
				}
			}
		}
	}
}
extern "C"
void cuda_fluid_initial(void *velocityinput, float3 size_WHD, size_t pitch, size_t pitch_slice, float value){
	hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((size_WHD.x+Db.x-1)/Db.x, (size_WHD.y+Db.y-1)/Db.y);

    cuda_kernel_initial<<<Dg,Db>>>((unsigned char *)velocityinput, size_WHD, pitch, pitch_slice, value);

    error = hipGetLastError();
    if (error != hipSuccess){
        printf("cuda_kernel_initial() failed to launch error = %d\n", error);
    }
}