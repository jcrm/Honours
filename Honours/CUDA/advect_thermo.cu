#include "hip/hip_runtime.h"
#ifndef _ADVECT_THERMO_CUDA_
#define _ADVECT_THERMO_CUDA_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""

#include "../Source/CUDA/cuda_header.h"

__global__ void cuda_kernel_advect_thermo(float *input, float *velocity, Size size){ 
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){ 
		if((x_iter - 1 >= 0 && x_iter + 1 < size.width_) && (y_iter - 1 >= 0 && y_iter + 1 < size.height_) && (z_iter - 1 >= 0 && z_iter + 1 < size.depth_)){
			float *cellVelo = velocity + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
			float3 pos = {x_iter, y_iter, z_iter};
			float3 cell_velo = {cellVelo[x_identifier_], cellVelo[y_identifier_], cellVelo[z_identifier_]};
			pos.x = pos.x - (time_step * cell_velo.x);
			pos.y = pos.y - (time_step * cell_velo.y);
			pos.z = pos.z - (time_step * cell_velo.z);
			
			int3 location = {pos.x,pos.y, pos.z};
			if(location.x < 0){
				location.x = 0;
			}
			if(location.y <0){
				location.y = 0;
			}
			if(location.z < 0){
				location.z = 0;
			}
			if(location.x >= size.width_){
				location.x = size.width_ - 1;
			}
			if(location.y >= size.height_){
				location.y = size.height_ - 1;
			}
			if(location.z >= size.depth_){
				location.z = size.depth_ - 1;
			}

			int3 location_two = {location.x+1,location.y+1, location.z+1};
			if(location_two.x >= size.width_){
				location_two.x = size.width_ - 1;
			}
			if(location_two.y >= size.height_){
				location_two.y = size.height_ - 1;
			}
			if(location_two.z >= size.depth_){
				location_two.z = size.depth_ - 1;
			}
			float *field_left_up = input + (location.z*size.pitch_slice_) + (location.y*size.pitch_) + (PIXEL_FMT_SIZE_RG * location.x);
			float *field_left_down = input + (location.z*size.pitch_slice_) + (location_two.y*size.pitch_) + (PIXEL_FMT_SIZE_RG * location.x);
			float *field_right_up = input + (location.z*size.pitch_slice_) + (location.y*size.pitch_) + (PIXEL_FMT_SIZE_RG * location_two.x);
			float *field_right_down = input + (location.z*size.pitch_slice_) + (location_two.y*size.pitch_) + (PIXEL_FMT_SIZE_RG * location_two.x);

			float *field_left_up_back = input + (location_two.z*size.pitch_slice_) + (location.y*size.pitch_) + (PIXEL_FMT_SIZE_RG * location.x);
			float *field_left_down_back = input + (location_two.z*size.pitch_slice_) + (location_two.y*size.pitch_) + (PIXEL_FMT_SIZE_RG * location.x);
			float *field_right_up_back = input + (location_two.z*size.pitch_slice_) + (location.y*size.pitch_) + (PIXEL_FMT_SIZE_RG * location_two.x);
			float *field_right_down_back = input + (location_two.z*size.pitch_slice_) + (location_two.y*size.pitch_) + (PIXEL_FMT_SIZE_RG * location_two.x);

			float temp_1 = field_left_up[theta_identifier_] + field_left_down[theta_identifier_] +  field_right_up[theta_identifier_] + field_right_down[theta_identifier_];
			float temp_2 = field_left_up_back[theta_identifier_] + field_left_down_back[theta_identifier_] + field_right_down_back[theta_identifier_] + field_right_up_back[theta_identifier_];

			temp_1 /=4.f;
			temp_2 /=4.f;
			
			float*output_thermo = input + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RG * x_iter);
			output_thermo[theta_advect_identifier_] = (temp_1 + temp_2)/2.f;
		}
	}
}

extern "C"
void cuda_fluid_advect_thermo(void *input, void* velocity, Size size){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_advect_thermo<<<Dg,Db>>>((float*)input, (float*)velocity, size);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_kernel_advect() failed to launch error = %d\n", error);
	}
}
#endif