#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

__global__ void cuda_kernel_project(unsigned char *pressure, unsigned char* velocityInput,float3 sizeWHD, size_t pitch, size_t pitchSlice){
	int xIter = blockIdx.x*blockDim.x + threadIdx.x;
	int yIter = blockIdx.y*blockDim.y + threadIdx.y;
	int zIter = 0;

	 // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (xIter >= sizeWHD.x || yIter >= sizeWHD.y) return;

	for(zIter = 0; zIter < sizeWHD.z; ++zIter){ 
		// Get pressure values from neighboring cells. 
		unsigned char *pLeft = NULL;
		unsigned char *pRight = NULL;
		unsigned char *pDown = NULL;
		unsigned char *pUp = NULL;
		unsigned char *pBottom = NULL;
		unsigned char *pTop = NULL;

		if(xIter - 1 < 0){
			pLeft = pressure + (zIter*pitchSlice) + (yIter*pitch) + (4*xIter);
		}else{
			pLeft = pressure + (zIter*pitchSlice) + (yIter*pitch) + (4*(xIter-1));
		}
		if(xIter + 1 ==sizeWHD.x){
			pRight = pressure + (zIter*pitchSlice) + (yIter*pitch) + (4*xIter);
		}else{
			pRight = pressure + (zIter*pitchSlice) + (yIter*pitch) + (4*(xIter+1));
		}

		if(yIter - 1 < 0){
			pDown = pressure + (zIter*pitchSlice) + (yIter*pitch) + (4*xIter); 
		}else{
			pDown = pressure + (zIter*pitchSlice) + ((yIter-1)*pitch) + (4*xIter); 
		}
		if(yIter + 1 ==sizeWHD.y){
			pUp = pressure + (zIter*pitchSlice) + (yIter*pitch) + (4*xIter); 
		}else{
			pUp = pressure + (zIter*pitchSlice) + ((yIter+1)*pitch) + (4*xIter); 
		}

		if(zIter - 1 < 0){
			pTop = pressure + (zIter*pitchSlice) + (yIter*pitch) + (4*xIter);
		}else{
			pTop = pressure + ((zIter-1)*pitchSlice) + (yIter*pitch) + (4*xIter);
		}
		if(zIter + 1 ==sizeWHD.y){
			pBottom = pressure + (zIter*pitchSlice) + (yIter*pitch) + (4*xIter);
		}else{
			pBottom = pressure + ((zIter+1)*pitchSlice) + (yIter*pitch) + (4*xIter);
		}
		float3 gradP;
		gradP.x = 0.5 *(pRight[0] - pLeft[0]);
		gradP.y = 0.5 *(pTop[0] - pBottom[0]);
		gradP.z = 0.5 *(pUp[0] - pDown[0]);
		unsigned char* cellVelocity = velocityInput + (zIter*pitchSlice) + (yIter*pitch) + (4*xIter);
		// Project the velocity onto its divergence-free component by  
		// subtracting the gradient of pressure.  
		float3 vOld;  
		vOld.x = cellVelocity[0];
		vOld.y = cellVelocity[1];
		vOld.z = cellVelocity[2];
		cellVelocity[0] = vOld.x - gradP.x;
		cellVelocity[1]= vOld.y - gradP.y;
		cellVelocity[2] = vOld.z - gradP.z; 
	}
}

extern "C"
void cuda_fluid_project(void *pressure, void *velocityInput, float3 sizeWHD, size_t pitch, size_t pitchSlice){
	hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((sizeWHD.x+Db.x-1)/Db.x, (sizeWHD.y+Db.y-1)/Db.y);

    cuda_kernel_project<<<Dg,Db>>>((unsigned char *)pressure, (unsigned char *)velocityInput,sizeWHD, pitch, pitchSlice);

    error = hipGetLastError();
    if (error != hipSuccess){
        printf("cuda_kernel_project() failed to launch error = %d\n", error);
    }
}