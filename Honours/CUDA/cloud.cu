#include "hip/hip_runtime.h"
#ifndef _CLOUD_CUDA_
#define _CLOUD_CUDA_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""
#include "math.h"

#include "../Source/CUDA/cuda_header.h"

//output velocity derrivitive teture //input velcoity texutre
__global__ void cuda_kernel_project(float*pressure, float* velocity, float* advect, Size size){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; z_iter++){ 
		float*cell_velocity = velocity + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
		float*cell_velocity_advect = advect + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);

		if(x_iter +1 < size.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){
					// Get pressure values from neighboring cells. 
					float*pLeft = pressure + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * (x_iter-1));
					float*pRight = pressure + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * (x_iter+1));
					float*pDown = pressure + (z_iter*size.pitch_slice_) + ((y_iter-1)*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter); 
					float*pUp = pressure + (z_iter*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter); 
					float*pTop = pressure + ((z_iter-1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
					float*pBottom = pressure + ((z_iter+1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);

					float fr = pRight[pressure_identifier_];
					float fl = pLeft[pressure_identifier_];
					float ft = pTop[pressure_identifier_];
					float fb = pBottom[pressure_identifier_];
					float fu = pUp[pressure_identifier_];
					float fd = pDown[pressure_identifier_];

					float temp_x = (fr - fl);
					float temp_y = (fu - fd);
					float temp_z = (fb - ft);

					float new_x = cell_velocity_advect[x_identifier_];
					float new_y = cell_velocity_advect[y_identifier_];
					float new_z = cell_velocity_advect[z_identifier_];
					
					new_x = new_x - 0.5f*(time_step * temp_x);
					new_y = new_y - 0.5f*(time_step * temp_y);
					new_z = new_z - 0.5f*(time_step * temp_z);
					
					cell_velocity[x_identifier_] = new_x;
					cell_velocity[y_identifier_] = new_y;
					cell_velocity[z_identifier_] = new_z; 

					float density = (cell_velocity[x_identifier_] * cell_velocity[x_identifier_]) + 
						(cell_velocity[y_identifier_] * cell_velocity[y_identifier_]) + 
						(cell_velocity[z_identifier_] * cell_velocity[z_identifier_]);
					density = sqrt(density);
					//density =0.5f;
					cell_velocity[3] = density;
				}
			}
		}
	}
}

extern "C"
void cuda_fluid_project(void *pressure, void *velocityInput, void* advect, Size size){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_project<<<Dg,Db>>>((float *)pressure, (float *)velocityInput, (float*)advect, size);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_fluid_project() failed to launch error = %d\n", error);
	}
}
#endif