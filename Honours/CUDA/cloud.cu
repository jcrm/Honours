#include "hip/hip_runtime.h"
#ifndef _CLOUD_CUDA_
#define _CLOUD_CUDA_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include ""
#include "math.h"

#include "../Source/CUDA/cuda_header.h"

//output velocity derrivitive teture //input velcoity texutre
__global__ void cuda_kernel_project(float*pressure, float* velocity, Size size, int pressure_index){
	int x_iter = blockIdx.x*blockDim.x + threadIdx.x;
	int y_iter = blockIdx.y*blockDim.y + threadIdx.y;
	int z_iter = 0;

	for(z_iter = 0; z_iter < size.depth_; ++z_iter){ 
		float*cell_velocity = velocity + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
		if(x_iter +1 < size.width_ && x_iter - 1 >= 0){
			if(y_iter + 1 < size.height_ && y_iter - 1 >= 0){
				if(z_iter + 1 < size.depth_ && z_iter - 1 >= 0){
					// Get pressure values from neighboring cells. 
					float*pLeft = pressure + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * (x_iter-1));
					float*pRight = pressure + (z_iter*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * (x_iter+1));
					float*pDown = pressure + (z_iter*size.pitch_slice_) + ((y_iter-1)*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter); 
					float*pUp = pressure + (z_iter*size.pitch_slice_) + ((y_iter+1)*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter); 
					float*pTop = pressure + ((z_iter-1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);
					float*pBottom = pressure + ((z_iter+1)*size.pitch_slice_) + (y_iter*size.pitch_) + (PIXEL_FMT_SIZE_RGBA * x_iter);

					float temp_x = (-pRight[pressure_index]) - pLeft[pressure_index];
					float temp_y = (-pTop[pressure_index]) - pBottom[pressure_index];
					float temp_z = (-pUp[pressure_index]) - pDown[pressure_index];

					float new_x = cell_velocity[x_identifier_];
					float new_y = cell_velocity[y_identifier_];
					float new_z = cell_velocity[z_identifier_];

					new_x = new_x - (time_step * temp_x);
					new_y = new_y - (time_step * temp_y);
					new_z = new_z - (time_step * temp_z);

					cell_velocity[x_identifier_] = new_x;
					cell_velocity[y_identifier_] = new_y;
					cell_velocity[z_identifier_] = new_z; 

					float density = (cell_velocity[x_identifier_] * cell_velocity[x_identifier_]) + 
						(cell_velocity[y_identifier_] * cell_velocity[y_identifier_]) + 
						(cell_velocity[z_identifier_] * cell_velocity[z_identifier_]);
					density = sqrt(density);
					//density =0.5f;
					cell_velocity[3] = density;
				}
			}
		}
	}
}

extern "C"
void cuda_fluid_project(void *pressure, void *velocityInput, Size size, int pressure_index){
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((size.width_+Db.x-1)/Db.x, (size.height_+Db.y-1)/Db.y);

	cuda_kernel_project<<<Dg,Db>>>((float *)pressure, (float *)velocityInput, size, pressure_index);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("cuda_fluid_project() failed to launch error = %d\n", error);
	}
}
#endif