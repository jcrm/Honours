#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

__global__ void cuda_kernel_project(unsigned char *pressure, unsigned char* velocityInput,float3 size_WHD, size_t pitch, size_t pitch_slice, int pressure_index){
	int xIter = blockIdx.x*blockDim.x + threadIdx.x;
	int yIter = blockIdx.y*blockDim.y + threadIdx.y;
	int zIter = 0;

	for(zIter = 0; zIter < size_WHD.z; ++zIter){ 
		if(xIter +1 < size_WHD.x){
			if(xIter - 1 > 0){
				if(yIter + 1 < size_WHD.y){
					if(yIter - 1 > 0){
						if(zIter + 1 < size_WHD.z){
							if(zIter - 1 > 0){
								// Get pressure values from neighboring cells. 
								unsigned char *pLeft = pressure + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter-1));
								unsigned char *pRight = pressure + (zIter*pitch_slice) + (yIter*pitch) + (4*(xIter+1));
								unsigned char *pDown = pressure + (zIter*pitch_slice) + ((yIter-1)*pitch) + (4*xIter); 
								unsigned char *pUp = pressure + (zIter*pitch_slice) + ((yIter+1)*pitch) + (4*xIter); 
								unsigned char *pTop = pressure + ((zIter-1)*pitch_slice) + (yIter*pitch) + (4*xIter);
								unsigned char *pBottom = pressure + ((zIter+1)*pitch_slice) + (yIter*pitch) + (4*xIter);
								unsigned char* cellVelocity = velocityInput + (zIter*pitch_slice) + (yIter*pitch) + (4*xIter);
								cellVelocity[0] = cellVelocity[0] - (0.5f *(pRight[pressure_index] - pLeft[pressure_index]));
								cellVelocity[1]= cellVelocity[1] - (0.5f *(pTop[pressure_index] - pBottom[pressure_index]));
								cellVelocity[2] = cellVelocity[2] - (0.5f *(pUp[pressure_index] - pDown[pressure_index])); 
							}
						}
					}
				}
			}
		}
		
	}
}

extern "C"
void cuda_fluid_project(void *pressure, void *velocityInput, float3 size_WHD, size_t pitch, size_t pitch_slice, int pressure_index){
	hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((size_WHD.x+Db.x-1)/Db.x, (size_WHD.y+Db.y-1)/Db.y);

    cuda_kernel_project<<<Dg,Db>>>((unsigned char *)pressure, (unsigned char *)velocityInput,size_WHD, pitch, pitch_slice, pressure_index);

    error = hipGetLastError();
    if (error != hipSuccess){
        printf("cuda_fluid_project() failed to launch error = %d\n", error);
    }
}